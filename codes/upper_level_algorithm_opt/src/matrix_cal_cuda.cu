#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <chrono>
#include <cstring>
#include "read_data.hpp"
#include "hip/hip_runtime.h"

__global__ void matrixMultiply(double *m1, double* m2, double *answer, size_t m, size_t n, size_t p) {
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < p) {
        double sum = 0;
        for (size_t k = 0; k < n; k++) {
            sum += m1[row * n + k] * m2[k * p + col];
        }

        answer[row * p + col] = sum;
    }
}

int main(int argc, char* argv[]) {
    std::string folder = "./";
    if (argc >= 2) {
        folder = argv[1];
    }
    auto files = get_files(folder);
    if (files.has_value()) {
        for (const auto& level : files.value()) {
            auto msize = level.first;
            size_t m = std::get<0>(msize);
            size_t n = std::get<1>(msize);
            size_t p = std::get<2>(msize);
            std::cout << "测试矩阵规模：" << m << " " << n << " " << p << std::endl;
            std::cout << "每个数据测量 5 次" << std::endl;
            std::cout << "数据测量交替执行" << std::endl;
            for (int times = 0; times < 5; times++) {
                for (const auto& file : level.second) {
                    auto matrixs = get_matrixs(file);
                    if (matrixs.has_value()) {
                        auto tup = matrixs.value();
                        double* m1 = std::get<0>(tup);
                        double* m2 = std::get<1>(tup);
                        double* m3 = std::get<2>(tup);

                        double* answer = static_cast<double*>(operator new(m * p * sizeof(double)));
                        for (size_t init = 0; init < m * p; init++) {
                            answer[init] = 0;
                        }

                        double *dm1, *dm2, *danswer;
                        //============= CUDA =============
                        auto start = std::chrono::high_resolution_clock::now();

                        hipMalloc((void**)&dm1, m * n * sizeof(double));
                        hipMalloc((void**)&dm2, n * p * sizeof(double));
                        hipMalloc((void**)&danswer, m * p * sizeof(double));

                        hipMemcpy(dm1, m1, m * n * sizeof(double), hipMemcpyHostToDevice);
                        hipMemcpy(dm2, m2, n * p * sizeof(double), hipMemcpyHostToDevice);

                        dim3 grid((p + 15) / 16, (m + 15) / 16);
                        dim3 block(16, 16);

                        matrixMultiply<<<grid, block>>>(dm1, dm2, danswer, m, n, p);

                        hipMemcpy(answer, danswer, m * p * sizeof(double), hipMemcpyDeviceToHost);

                        auto end = std::chrono::high_resolution_clock::now();
                        //============= CUDA =============

                        // 验证
                        size_t ok = 0;
                        for (; ok < m * p; ok++) {
                            if (std::abs(answer[ok] - m3[ok]) >= EPSILON) {
                                break;
                            }
                        }

                        if (ok != m * p) {
                            std::cerr << file << " 第 " << times << " 次验证失败" << std::endl;
                            std::cerr << "出错位置：" << ok << std::endl;  
                            std::cerr << "output[" << ok << "] = " << answer[ok] << std::endl;
                            std::cerr << "answer[" << ok << "] = " << m3[ok] << std::endl;
                            print_matrix_data(m, n, m1);
                            print_matrix_data(n, p, m2);
                            print_matrix_data(m, p, m3);
                            print_matrix_data(m, p, answer);
                        } else {
                            auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
                            std::cout << file << " 第 " << times << " 次耗时: " << duration.count() / 1000000.0 << " 秒" << std::endl;
                        }

                        delete[] m1;
                        delete[] m2;
                        delete[] m3;
                        delete[] answer;
                        hipFree(dm1);
                        hipFree(dm2);
                        hipFree(danswer);
                    } else {
                        return EXIT_FAILURE;
                    }
                }
            }

        }
        std::cout << "测量结束" << std::endl;
    } else {
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}